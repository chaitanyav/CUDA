#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void handleError(hipError_t error, int lineno) {
  if (error != hipSuccess) {
    printf("Error: %s:%d\n", __FILE__, lineno);
    printf("Code: %d, Reason: %s\n", error, hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

void initializeInt(int *iptr, int size) {
  for(int i = 0; i < size; i++) {
    iptr[i] = i;
  }
}

void printMatrix(int *iptr, const int nx, const int ny) {
  int *C = iptr;
    for(int i = 0; i < nx; i++) {
      for(int j = 0; j < ny; j++) {
        printf("%3d\n", C[j]);
      }
        C += ny;
        printf("\n");
    }
}

__global__ void printThreadIndex(int *A, const int nx, const int ny) {
  int ix = threadIdx.x  + blockIdx.x * blockDim.x;
  int iy = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int idx = iy * nx + ix;

  printf("thread_id (%d, %d) block_id (%d, %d) coordinate (%d, %d) global index %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}

int main(int argc, char *argv[]) {
  int dev = 0;
  hipDeviceProp_t deviceProp;
  handleError(hipGetDeviceProperties(&deviceProp, dev), __LINE__);
  printf("Using device %d:%s\n", dev, deviceProp.name);
  handleError(hipSetDevice(dev), __LINE__);

  int nx = 8;
  int ny = 6;
  int nxy = nx * ny;
  int nBytes = nxy * sizeof(float);

  int *h_A;
  h_A = (int *)malloc(nBytes);
  initializeInt(h_A, nBytes);
  printMatrix(h_A, nx, ny);

  int *d_A;
  handleError(hipMalloc((void **)&d_A, nBytes), __LINE__);
  handleError(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice), __LINE__);

  dim3 block(4, 2);
  dim3 grid((nx + block.x - 1)/ block.x, (ny + block.y - 1)/ block.y);
  printThreadIndex<<<grid, block>>>(d_A, nx, ny);
  handleError(hipDeviceSynchronize(), __LINE__);
  hipFree(d_A);
  free(h_A);
  hipDeviceReset();
  return 0;
}
