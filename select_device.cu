
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void handleError(hipError_t error) {
  if(error != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char *argv[]) {
  hipDeviceProp_t prop;
  int dev;
  handleError(hipGetDevice(&dev));
  printf("Current Device ID is %d\n", dev);

  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.major = 3;
  prop.minor = 0;

  handleError(hipChooseDevice(&dev, &prop));
  printf("The closest device to revision 3.0 is %d\n", dev);
  handleError(hipSetDevice(dev));
  return 0;
}
