#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void handleError(hipError_t error, int lineno) {
  if(error != hipSuccess) {
    printf("Error: %s %d\n", __FILE__, lineno);
    printf("code: %d, reason %s\n", error, hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

void initializeData(float *ptr, int size) {
 time_t t;
 srand((unsigned) time(&t));
 for(int pos = 0; pos < size; pos++) {
   ptr[pos] = (float) (rand() & 0xFF) / 10.0F;
 }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C) {
  int id = threadIdx.x;
  C[id] = A[id] + B[id];
}

int main(int argc, char *argv[]) {
  int dev = 0;
  hipSetDevice(dev);

  int nElem = 1024;
  size_t nBytes = nElem * sizeof(float);
  float *h_A, *h_B, *h_C;
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  h_C = (float *)malloc(nBytes);
  initializeData(h_A, nElem);
  initializeData(h_B, nElem);
  float *d_A, *d_B, *d_C;
  handleError(hipMalloc((float **)&d_A, nBytes), __LINE__);
  handleError(hipMalloc((float **)&d_B, nBytes), __LINE__);
  handleError(hipMalloc((float **)&d_C, nBytes), __LINE__);

  handleError(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice), __LINE__);
  handleError(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice), __LINE__);

  dim3 block(nElem);
  dim3 grid(nElem/block.x);

  sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C);
  handleError(hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost),
      __LINE__);

  for(int pos = 0; pos < nElem; pos++) {
    printf("%f + %f = %f\n", h_A[pos], h_B[pos], h_C[pos]);
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);
  return 0;
}
