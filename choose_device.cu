/*
* Author: NagaChaitanya Vellanki
*
*
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void handleError(hipError_t err, const char *file, int line) {
 if(err != hipSuccess) {
 printf( "%s in %s at line %d\n", hipGetErrorString( err ),
				file, line);
		exit( EXIT_FAILURE );
 }
}


int main(int argc, char *argv[]) {
hipDeviceProp_t prop;
int device;

handleError(hipGetDevice(&device), __FILE__, __LINE__);
printf("Device is %d\n", device);

memset(&prop, 0, sizeof(hipDeviceProp_t));
prop.major = 2;
prop.minor = 1;
handleError(hipChooseDevice(&device, &prop), __FILE__, __LINE__);
printf("Chosen Device is %d\n", device);

handleError(hipSetDevice(device), __FILE__, __LINE__);

exit(EXIT_SUCCESS);
}