#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void add(int a, int b, int *c) {
*c = a + b;
}

void handleError(hipError_t err, const char *file, int line) {
 if(err != hipSuccess) {
 printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                __FILE__, __LINE__ );
        exit( EXIT_FAILURE );
 }
}

int main(int argc, char *argv[]) {
 int c;
 int *dev_c;

 handleError(hipMalloc((void **) &dev_c, sizeof(int)), __FILE__, __LINE__);

 add<<<1,1>>>(2,7, dev_c);

 handleError(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost), __FILE__, __LINE__);

 printf("2 + 7 = %d\n", c);
 hipFree(dev_c);

 exit(EXIT_SUCCESS);
}