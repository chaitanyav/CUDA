
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000

void handleError(hipError_t error) {
  if(error != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x;
    if(tid < N) {
      c[tid] = a[tid] + b[tid];
    }
}

int main(int argc, char *argv[]) {
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  handleError(hipMalloc((void **) &dev_a, sizeof(int) * N));
  handleError(hipMalloc((void **) &dev_b, sizeof(int) * N));
  handleError(hipMalloc((void **) &dev_c, sizeof(int) * N));

  for(int i = 0; i < N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }

  handleError(hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice));
  handleError(hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice));

  add<<<N,1>>>(dev_a, dev_b, dev_c);

  handleError(hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost));

  for(int i = 0; i <  N; i++) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  return 0;
}
