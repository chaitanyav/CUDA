
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define N 500

__global__ void add(int *a, int *b, int *c) {
  int tid = threadIdx.x;
  if(tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

void handleError(hipError_t error, int lineNo) {
  if(error != hipSuccess) {
    printf("Error: %s in %s at line %d\n", hipGetErrorString(error), __FILE__, lineNo);
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char *argv[]) {
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  handleError(hipMalloc((void **)&dev_a, N * sizeof(int)), __LINE__);
  handleError(hipMalloc((void **)&dev_b, N * sizeof(int)), __LINE__);
  handleError(hipMalloc((void **)&dev_c, N * sizeof(int)), __LINE__);

  for(int i = 0; i < N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }

  handleError(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice), __LINE__);
  handleError(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice), __LINE__);
  handleError(hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice), __LINE__);

  add<<<1,N>>>(dev_a, dev_b, dev_c);
  handleError(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost), __LINE__);
  for(int i = 0; i < N; i++) {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  return 0;
}
