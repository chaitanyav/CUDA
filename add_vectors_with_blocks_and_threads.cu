
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

#define N 33 * 1024

__global__ void add(int *a, int *b, int *c) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while(tid < N) {
    c[tid] = a[tid] + b[tid];
    tid += blockDim.x * gridDim.x;
  }
}

void handleError(hipError_t error, int lineNo) {
  if(error != hipSuccess) {
    printf("Error: %s in %s at line %d\n", hipGetErrorString(error), __FILE__, lineNo);
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char *argv[]) {
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  handleError(hipMalloc((void **)&dev_a, N * sizeof(int)), __LINE__);
  handleError(hipMalloc((void **)&dev_b, N * sizeof(int)), __LINE__);
  handleError(hipMalloc((void **)&dev_c, N * sizeof(int)), __LINE__);

  for(int i = 0; i < N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }

  handleError(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice), __LINE__);
  handleError(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice), __LINE__);
  handleError(hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice), __LINE__);

  add<<<264, 128>>>(dev_a, dev_b, dev_c);
  handleError(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost), __LINE__);

  bool success = true;
  for(int i = 0; i < N; i++) {
    if(a[i] + b[i] != c[i]) {
      printf("Error: %d + %d != %d\n", a[i], b[i], c[i]);
      success = false;
    }
  }
  if(success) {
    printf("Addition successful\n");
  }
  return 0;
}
